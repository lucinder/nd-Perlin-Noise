#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 5
#define M 10
#define CUDA_WARN(XXX) \
    if(XXX != hipSuccess){ \
    printf("CUDA Error: %s, at line %d\n", hipGetErrorString(XXX), __LINE__); }

__device__ int hash[256] = { 96, 131, 210, 231, 40, 179, 184, 56, 133, 209, 188, 207, 176, 245, 218, 230, 185, 70, 76, 105, 214, 182, 174, 72, 146, 159, 162, 14, 227, 160, 82, 212, 192, 191, 172, 74, 157, 236, 39, 26, 226, 201, 250, 211, 81, 254, 244, 219, 107, 161, 24, 53, 5, 154, 253, 34, 145, 197, 112, 233, 23, 68, 87, 49, 8, 156, 196, 248, 27, 149, 111, 19, 4, 62, 203, 190, 25, 132, 140, 202, 65, 16, 141, 118, 104, 153, 113, 144, 67, 175, 37, 216, 114, 60, 243, 189, 101, 150, 220, 217, 12, 252, 206, 124, 71, 103, 69, 171, 38, 126, 152, 167, 121, 178, 78, 106, 86, 15, 194, 66, 10, 237, 99, 55, 77, 13, 57, 205, 30, 44, 89, 138, 88, 41, 187, 73, 241, 221, 92, 215, 125, 168, 1, 46, 29, 239, 193, 52, 143, 251, 128, 61, 129, 45, 242, 64, 63, 213, 0, 123, 238, 43, 35, 208, 22, 33, 169, 222, 50, 51, 59, 32, 83, 20, 180, 183, 17, 108, 198, 177, 18, 80, 199, 94, 3, 9, 2, 170, 130, 186, 95, 165, 247, 204, 142, 28, 229, 102, 195, 116, 224, 163, 164, 97, 47, 36, 31, 223, 151, 225, 100, 122, 135, 136, 109, 84, 166, 249, 119, 7, 246, 155, 120, 235, 200, 181, 255, 127, 147, 21, 137, 58, 42, 75, 228, 54, 90, 232, 85, 93, 6, 79, 117, 98, 134, 173, 91, 148, 234, 240, 158, 11, 110, 48, 139, 115 };

__global__ void perlin(float* dnoise)
{
    // printf("Called function\n");
    int i = blockIdx.x * blockDim.x + threadIdx.x; // current point
    if (i > pow(M, N)) {
        // printf("Returning from OOB index %d\n",i);
        return;
    }
    dnoise[i] = 0.0f;
    float coords[N]; // coordinate array
    hiprandState state;
    hiprand_init(i, 0, 0, &state); // initiate using current thread
    for (int j = 0; j < N; j++) {
        coords[j] = 0.0f;
        coords[j] = hiprand_uniform(&state); // fill with random values
        // printf("%f", coords[j]);
    }
    float* gradients = (float*)malloc(sizeof(float) * pow(2, N)); // gradient vector
    int seed = hash[i % 256]; // reseed with pseudorandomness
    hiprand_init(seed, 0, 0, &state); // reseed with hashing
    for (int j = 0; j < pow(2, N); j++) {
        float euclid_dist = 0;
        float curgrad[N];
        for (int k = 0; k < N; k++) {
            curgrad[k] = 0; // initialize to 0 for safety
            curgrad[k] = hiprand_uniform(&state) * 2.0f - 1.0f; // load pseudorandom gradient
            // printf("%f ", curgrad[k]);
            euclid_dist += pow(curgrad[k], 2); // add to sum of squares
        }
        // printf("\n");
        // printf("%f ", gradients[j * N]);
        euclid_dist = sqrt(euclid_dist); // get euclidean distance
        int bin = ((j & (1 << (-1))) >> (-1));
        float dist = coords[0] - (float)bin;
        // printf("%f ", dist);
        gradients[j] = (curgrad[0] / euclid_dist) * dist;
        // printf("%f ", gradients[j * N]);
        for (int k = 1; k < N; k++) {
            bin = ((j & (1 << (k - 1))) >> (k - 1));
            dist = coords[k] - (float)bin;
            // printf("%d ", bin);
            gradients[j] += (curgrad[k] / euclid_dist) * dist; // calculate distance + dot product, add to dot product indices
        }
        // printf("%f ", gradients[j * N]);
    }
    // linear interpolation
    int step = 1;
    int dim = 0; // what dimension were looking at
    while (step < pow(2, N)) {
        for (int j = 0; j <= pow(2, N); j += 2 * step) {
            float f = coords[dim];
            f = f * f * f * (f * (f * 6 - 15) + 10); // apply fade function
            gradients[j] = gradients[j] + f * ((gradients[j + step]) - (gradients[j])); // interpolate along dim'th dimension
        }
        dim++;
        step *= 2;
    }
    // in theory gradients[0] should have our final value
    // printf("%f\n", gradients[0]);
    dnoise[i] = gradients[0];
    free(gradients);

    if (dnoise[i] == 0.0 || dnoise[i] < -10.0) {
        printf("Dangerous value detected at thread %d\n", i);
    }

    // printf("%f\n", dnoise[i]);
    // printf("Completed noisegen at index %d\n", i);
}

int perlin_parallel() {
    printf("Matrix size: %d^%d\n", M, N);
    if (pow(INT_MAX, 1.0 / (N)) * pow(512, 1.0/(N)) < M) {
        printf("ERROR: matrix size out of bounds!\n");
        return 1;
    }
    hipSetDevice(0);
    hipEvent_t start, stop;
    hipEvent_t start2, stop2;
    float elapsedTime, elapsedTimeWithMem;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    const int mSize = pow(M, N);
    const int mBytes = mSize * sizeof(float);
    printf("Bytes: %d\n", mBytes);

    hipEventRecord(start, 0);
    float* noise = (float*)malloc(mBytes);
    float* dev_noise = 0;
    CUDA_WARN(hipMalloc((void**)&dev_noise, mBytes));

    int n_thr = min(512, M); // if we can do our whole matrix in 1 block, do it, otherwise use 1024 thr/block
    int n_blk = (mSize / n_thr);
    if ((pow(M, N) / (float)n_thr) != (int)(pow(M, N) / (float)n_thr)) {
        printf("Adding kernel space for non-32-multiple matrix\n");
        n_blk++;
    }
    printf("Kernel configuration: %d blocks, %d threads.\n", n_blk, n_thr);

    hipEventRecord(start2, 0);
    perlin <<<n_blk, n_thr >> > (dev_noise);
    hipDeviceSynchronize();
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsedTime, start2, stop2);
    printf("Runtime (noise generation only): %.7f seconds\n", elapsedTime / 1000.0);

    CUDA_WARN(hipMemcpy(noise, dev_noise, mBytes, hipMemcpyDeviceToHost));
    /*
    for (int i = 0; i < pow(m, N); i++) {
        printf("%f\n", noise[i]);
    }
    */
    // printf("Noise generation complete. Loading noise to file.\n");
    
    FILE* fptr;
    fptr = fopen("perlin_out.txt", "w+");
    fprintf(fptr, "%d,%d\n",M,N);
    bool warn_flag = true;
    for (int i = 0; i < mSize; i++) {
        if ((noise[i] <= -10.0 || noise[i] == 0.0) && warn_flag) {
            printf("WARNING: Noise values may be incorrect.\n");
            warn_flag = false;
        }
        fprintf(fptr,"%f\n",noise[i]);
    }
    
    hipFree(dev_noise);
    free(noise);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTimeWithMem, start, stop);
    printf("Runtime with memory + filewrite operations: %.7f seconds\n", elapsedTimeWithMem / 1000.0);
    return 0;
}

// Helper method to get a timestamp - mostly courtesy of Gabriel Staples from https://stackoverflow.com/a/67731965
uint64_t GetTimeStamp() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return (((uint64_t)ts.tv_sec) * 1000000000) + ((uint64_t)ts.tv_nsec);
}

int perlin_sequential() {
    uint64_t timeStart, timeStop;

    const int mSize = pow(M, N);
    const int mBytes = mSize * sizeof(float);
    float* noise = (float*)malloc(mBytes);
    float* grid = (float*)malloc(N*mBytes);
    float* gradients = (float*)malloc(sizeof(float) * pow(2, N)); // gradient vector
    printf("Matrix size: %d^%d\n", M, N);
    printf("Bytes: %d\n", mBytes);
    srand(time(NULL));
    for (int i = 0; i <mSize*N; i++) {
        grid[i] = (float)rand() / (float)RAND_MAX; // fill with random values
        // printf("%f", grid[i]);
    }

    timeStart = GetTimeStamp();
    for (int i = 0; i < mSize; i++) {
        srand(i % 256);
        noise[i] = 0.0f;
        for (int j = 0; j < pow(2, N); j++) {
            float euclid_dist = 0;
            float curgrad[N];
            for (int k = 0; k < N; k++) {
                curgrad[k] = ((float)rand() / (float)(RAND_MAX)) * 2.0f - 1.0f; // load pseudorandom gradient
                // printf("%f ", curgrad[k]);
                euclid_dist += pow(curgrad[k], 2); // add to sum of squares
            }
            // printf("\n");
            // printf("%f ", gradients[j * N]);
            euclid_dist = sqrt(euclid_dist); // get euclidean distance
            int bin = ((j & (1 << (-1))) >> (-1));
            float dist = grid[i*N] - (float)bin;
            // printf("%f ", dist);
            gradients[j] = (curgrad[0] / euclid_dist) * dist;
            // printf("%f ", gradients[j * N]);
            for (int k = 1; k < N; k++) {
                bin = ((j & (1 << (k - 1))) >> (k - 1));
                dist = grid[i*N+k] - (float)bin;
                // printf("%d ", bin);
                gradients[j] += (curgrad[k] / euclid_dist) * dist; // calculate distance + dot product, add to dot product indices
            }
            // printf("%f ", gradients[j]);
        }
        // linear interpolation
        int step = 1;
        int dim = 0; // what dimension were looking at
        while (step < pow(2, N)) {
            for (int j = 0; j <= pow(2, N); j += 2 * step) {
                float f = grid[i*N+dim];
                f = f * f * f * (f * (f * 6 - 15) + 10); // apply fade function
                gradients[j] = gradients[j] + f * ((gradients[j + step]) - (gradients[j])); // interpolate along dim'th dimension
            }
            dim++;
            step *= 2;
        }
        // in theory gradients[0] should have our final value
        // printf("%f\n", gradients[0]);
        noise[i] = gradients[0];
        if (noise[i] == 0.0 || noise[i] < -10.0) {
            printf("Dangerous value detected at index %d: %f\n", i,noise[i]);
        }
    }
    timeStop = GetTimeStamp();
    float t_ms = (float)(timeStop - timeStart) / 1000.0;
    printf("Runtime (sequential): %.7f seconds\n", t_ms / 1000000.0);
    // free(gradients);
    free(noise);
    free(grid);
    return 0;
}

int main()
{
    // perlin_parallel();
    perlin_sequential();
    return 0;
}