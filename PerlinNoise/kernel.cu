#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 2
#define atoa(x) #x
__device__ int hash[256] = { 96, 131, 210, 231, 40, 179, 184, 56, 133, 209, 188, 207, 176, 245, 218, 230, 185, 70, 76, 105, 214, 182, 174, 72, 146, 159, 162, 14, 227, 160, 82, 212, 192, 191, 172, 74, 157, 236, 39, 26, 226, 201, 250, 211, 81, 254, 244, 219, 107, 161, 24, 53, 5, 154, 253, 34, 145, 197, 112, 233, 23, 68, 87, 49, 8, 156, 196, 248, 27, 149, 111, 19, 4, 62, 203, 190, 25, 132, 140, 202, 65, 16, 141, 118, 104, 153, 113, 144, 67, 175, 37, 216, 114, 60, 243, 189, 101, 150, 220, 217, 12, 252, 206, 124, 71, 103, 69, 171, 38, 126, 152, 167, 121, 178, 78, 106, 86, 15, 194, 66, 10, 237, 99, 55, 77, 13, 57, 205, 30, 44, 89, 138, 88, 41, 187, 73, 241, 221, 92, 215, 125, 168, 1, 46, 29, 239, 193, 52, 143, 251, 128, 61, 129, 45, 242, 64, 63, 213, 0, 123, 238, 43, 35, 208, 22, 33, 169, 222, 50, 51, 59, 32, 83, 20, 180, 183, 17, 108, 198, 177, 18, 80, 199, 94, 3, 9, 2, 170, 130, 186, 95, 165, 247, 204, 142, 28, 229, 102, 195, 116, 224, 163, 164, 97, 47, 36, 31, 223, 151, 225, 100, 122, 135, 136, 109, 84, 166, 249, 119, 7, 246, 155, 120, 235, 200, 181, 255, 127, 147, 21, 137, 58, 42, 75, 228, 54, 90, 232, 85, 93, 6, 79, 117, 98, 134, 173, 91, 148, 234, 240, 158, 11, 110, 48, 139, 115};

__global__ void perlin(float* noise, int* gradients)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int gradI = N*(hash[i % 256] % (sizeof(gradients)/N));
    // printf("Gradient index: %d\n", gradI);
    int currentGrad[N];
    for (int j = 0; j < N; j++) {
        currentGrad[j] = gradients[gradI + j]; // load gradients into current grad arr
        // printf("%d ", currentGrad[j]);
    }
    // printf("\n");
}

int* loadGradients() {
    int lineCount = 0;
    char pre[] = "gradients/gradient-";
    char suf[] = ".csv";
    char dim[2];
    sprintf(dim, "%d", N);
    char fileName[25];
    sprintf(fileName, "%s%s%s", pre, dim, suf);
    FILE* grad = fopen(fileName, "r");
    if (grad == NULL) {
        printf("File not found: %s\n",fileName);
        return 0;
    }
    else {
        printf("Loaded file %s\n", fileName);
    }
    char buffer[80];

    while (fgets(buffer, sizeof(buffer), grad) != NULL) { // count lines in gradient file
        lineCount++;
    }
    
    int* grads = (int*)malloc(lineCount * N * sizeof(int));

    int i = 0;
    while (fgets(buffer, 80, grad)) {
        char* token = strtok(buffer, ","); // read start of line
        while (token) {
            int n = atoi(token);
            // printf("%d\n", n);
            grads[i] = n;
            token = strtok(NULL, ",");
            i++;
        }
    }
    return grads;
}

int main()
{
    const int m = 10;
    float* noise = (float*)malloc(pow(m,N)*sizeof(float));
    float* dev_noise = 0;
    int* gradients = 0;
    int* host_gradients = loadGradients();
    hipMalloc((void**)&dev_noise, pow(m,N) * sizeof(float));
    hipMalloc((void**)&gradients, sizeof(host_gradients));

    hipMemcpy(gradients, host_gradients, sizeof(host_gradients), hipMemcpyHostToDevice);

    int n_thr = min(1024, m + (32 - (m % 32))); // round to nearest multiple of 32
    int n_blk = pow(m, N) / n_thr;
    perlin<<<n_blk, n_thr>>> (dev_noise, gradients);

    hipFree(dev_noise);
    // free(host_gradients);
    free(noise);
    return 0;
}